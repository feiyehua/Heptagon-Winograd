#include "hip/hip_runtime.h"
#include "output_transform.cuh"

__device__ inline tile_index_t device_get_tile_index(int64_t tile, tiling_info_t ts) {
  tile_index_t ti;
  ti.b = tile / ts.num_tile_per_image;
  tile = tile % ts.num_tile_per_image;
  ti.th = tile / ts.tiles_on_w;
  ti.tw = tile % ts.tiles_on_w;
  return ti;
}

__global__ void output_transform(hipPitchedPtr M,  // input tensor
                                 float* __restrict__ device_out_tensor,
                                 const tiling_info_t ti,
                                 const out_shape_t os,
                                 const int64_t collapsed_dim_size) {
  float* M_tensor = (float*)M.ptr;
  // float* Y_tensor = (float*)Y.ptr;
  float z0, z1, z2, z3, z4;
  int64_t idx = (blockIdx.x * blockDim.x + threadIdx.x);
  int64_t M_tensor_z = M.pitch / sizeof(float);
  int64_t M_tensor_yz = M_tensor_z * M.ysize;

  // int64_t Y_tensor_z = Y.pitch / sizeof(float);
  // int64_t Y_tensor_yz = Y.ysize * Y_tensor_z;

  if (idx >= collapsed_dim_size) {
    return;
  }
  /*
  AT =
  ⎡1  1  1   1  1   0⎤
  ⎢                  ⎥
  ⎢0  1  -1  2  -2  0⎥
  ⎢                  ⎥
  ⎢0  1  1   4  4   0⎥
  ⎢                  ⎥
  ⎣0  1  -1  8  -8  1⎦
  */
  float tmp_Y_tensor[4][6]={0};
#pragma unroll
  for (int64_t w = 0; w < ti.tile_in_w; ++w) {
    z4 = M_tensor[0 * M_tensor_yz + w * M_tensor_z + idx];
    z0 = z4;

    z4 = M_tensor[1 * M_tensor_yz + w * M_tensor_z + idx];
    z0 = z0 + z4;
    z1 = z4;
    z2 = z4;
    z3 = z4;

    z4 = M_tensor[2 * M_tensor_yz + w * M_tensor_z + idx];
    z0 += z4;
    z1 += -z4;
    z2 += z4;
    z3 += -z4;

    z4 = M_tensor[3 * M_tensor_yz + w * M_tensor_z + idx];
    z0 += z4;
    z1 += 2.0f * z4;
    z2 += 4.0f * z4;
    z3 += 8.0f * z4;

    z4 = M_tensor[4 * M_tensor_yz + w * M_tensor_z + idx];
    z0 += z4;
    z1 += -2.0f * z4;
    z2 += 4.0f * z4;
    z3 += -8.0f * z4;

    z4 = M_tensor[5 * M_tensor_yz + w * M_tensor_z + idx];
    z3 += z4;

    tmp_Y_tensor[0][w] = z0;
    tmp_Y_tensor[1][w] = z1;
    tmp_Y_tensor[2][w] = z2;
    tmp_Y_tensor[3][w] = z3;
    // Y_tensor[0 * Y_tensor_z + w + idx * Y_tensor_yz] = z0;
    // Y_tensor[1 * Y_tensor_z + w + idx * Y_tensor_yz] = z1;
    // Y_tensor[2 * Y_tensor_z + w + idx * Y_tensor_yz] = z2;
    // Y_tensor[3 * Y_tensor_z + w + idx * Y_tensor_yz] = z3;
  }
#pragma unroll
  for (int64_t h = 0; h < ti.tile_out_h; ++h) {
    z4 = tmp_Y_tensor[h][0];

    z0 = z4;

    z4 = tmp_Y_tensor[h][1];
    z0 += z4;
    z1 = z4;
    z2 = z4;
    z3 = z4;

    z4 = tmp_Y_tensor[h][2];
    z0 += z4;
    z1 += -z4;
    z2 += z4;
    z3 += -z4;

    z4 = tmp_Y_tensor[h][3];
    z0 += z4;
    z1 += 2.0f * z4;
    z2 += 4.0f * z4;
    z3 += 8.0f * z4;

    z4 = tmp_Y_tensor[h][4];
    z0 += z4;
    z1 += -2.0f * z4;
    z2 += 4.0f * z4;
    z3 += -8.0f * z4;

    z4 = tmp_Y_tensor[h][5];

    z3 += z4;

    tmp_Y_tensor[h][0] = z0;
    tmp_Y_tensor[h][1] = z1;
    tmp_Y_tensor[h][2] = z2;
    tmp_Y_tensor[h][3] = z3;
  }

  int64_t out_tensor_z = os.w;  // device_out_tensor.pitch / sizeof(float);
  int64_t out_tensor_yz = os.h * out_tensor_z;
  float* out_tensor = (float*)device_out_tensor;

  int64_t oc = idx / ti.num_tiles;
  int64_t tile = idx % ti.num_tiles;
  tile_index_t tidx = device_get_tile_index(tile, ti);
  int64_t batch = tidx.b, ww = tidx.tw, hh = tidx.th;
  if (batch >= os.bs) return;
#pragma unroll
  for (int64_t h = 0; h < min(ti.tile_out_h, os.h - 4 * hh); ++h) {
#pragma unroll
    for (int64_t w = 0; w < min(ti.tile_out_w, os.h - 4 * ww); ++w) {
      {
        if (hh * 4 + h < os.h && ww * 4 + w < os.w)
          out_tensor[(batch * os.oc + oc) * out_tensor_yz + (hh * 4 + h) * out_tensor_z +
                     (ww * 4 + w)] = tmp_Y_tensor[h][w];
      }
    }
  }
}

__global__ void device_output_unpacking_store(hipPitchedPtr device_Y_tensor,
                                              float* __restrict__ device_out_tensor,
                                              const out_shape_t os,
                                              const tiling_info_t ti) {
  // typedef float(*Y_tensor_t)[ti.num_tiles][ti.tile_in_h][ti.tile_out_w];
  // typedef float(*out_tensor_t)[os.oc][os.h][os.w];
  float* Y_tensor = (float*)device_Y_tensor.ptr;
  float* out_tensor = (float*)device_out_tensor;
  int64_t Y_tensor_z = device_Y_tensor.pitch / sizeof(float);
  int64_t Y_tensor_yz = device_Y_tensor.ysize * Y_tensor_z;

  int64_t out_tensor_z = os.w;  // device_out_tensor.pitch / sizeof(float);
  int64_t out_tensor_yz = os.h * out_tensor_z;

  int64_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  int64_t oc = tid % os.oc;
  int64_t tile = tid / os.oc;
  tile_index_t tidx = device_get_tile_index(tile, ti);
  int64_t batch = tidx.b, ww = tidx.tw, hh = tidx.th;
  if (batch >= os.bs) return;
#pragma unroll
  for (int64_t h = 0; h < min(ti.tile_out_h, os.h - 4 * hh); ++h) {
#pragma unroll
    for (int64_t w = 0; w < min(ti.tile_out_w, os.h - 4 * ww); ++w) {
      {
        if (hh * 4 + h < os.h && ww * 4 + w < os.w)
          out_tensor[(batch * os.oc + oc) * out_tensor_yz + (hh * 4 + h) * out_tensor_z +
                     (ww * 4 + w)] = Y_tensor[(oc * ti.num_tiles + tile) * Y_tensor_yz + h * Y_tensor_z + w];
      }
    }
  }
}

void device_output_transform(hipPitchedPtr device_M_tensor,          // input tensor
                             float* __restrict__ device_out1_tensor,  // output tensor
                             float* __restrict__ out,
                             const tiling_info_t ti,
                             const int64_t collapsed_dim_size,
                             const U_shape_t us,
                             const V_shape_t vs,
                             const out_shape_t os,
                             Device_Memory_Pool& device_Memory_Pool) {
  // 分配out_tensor内存
  hipPitchedPtr device_out_tensor;
  hipExtent device_out_tensor_extent = make_hipExtent(sizeof(float) * os.w, os.h, os.oc * os.bs);
  device_out_tensor.pitch = sizeof(float) * os.w;
  device_out_tensor.xsize = sizeof(float) * os.w;
  device_out_tensor.ysize = os.h;
  device_Memory_Pool.poolMalloc(&device_out_tensor.ptr, sizeof(float) * os.w * os.h * os.oc * os.bs);

  //计算out_tensor
  output_transform<<<DIV_UP(us.oc * vs.num_tiles, 128), 128>>>(
      device_M_tensor, (float*)device_out_tensor.ptr, ti, os, us.oc * vs.num_tiles);

  hipMemcpy(out, device_out_tensor.ptr, sizeof(float) * os.w * os.h * os.oc * os.bs, hipMemcpyDeviceToHost);
}