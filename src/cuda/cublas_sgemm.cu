#include <hipblas.h>
#include <stdio.h>

#include "utils.h"

void cublas_sgemm(hipblasHandle_t handle,
                  float* a,
                  int lda,
                  float* b,
                  int ldb,
                  float* c,
                  int ldc,
                  int m,
                  int n,
                  int k,
                  U_shape_t us,
                  V_shape_t vs,
                  tiling_info_t ti) {
  const float alpha = 1.f;
  const float beta = 0.f;
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  auto err = hipblasSgemm(handle, transa, transb, n, m, k, &alpha, b, ldb, a, lda, &beta, c, ldc);
  hipDeviceSynchronize();
}