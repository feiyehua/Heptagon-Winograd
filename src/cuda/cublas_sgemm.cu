#include <hipblas.h>
#include <stdio.h>

#include "utils.h"

void cublas_sgemm(hipblasHandle_t handle,
                  float* a,
                  int lda,
                  long long int strideA,
                  float* b,
                  int ldb,
                  long long int strideB,
                  float* c,
                  int ldc,
                  long long int strideC,
                  int m,
                  int n,
                  int k,
                  U_shape_t us,
                  V_shape_t vs,
                  tiling_info_t ti) {
  const float alpha = 1.f;
  const float beta = 0.f;
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  hipblasSgemmStridedBatched(handle,
                            transa,
                            transb,
                            n,
                            m,
                            k,
                            &alpha,
                            b,
                            ldb,
                            strideB,
                            a,
                            lda,
                            strideA,
                            &beta,
                            c,
                            ldc,
                            strideC,
                            ti.tile_in_h * ti.tile_in_w);
}