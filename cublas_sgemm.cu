#include <hipblas.h>
#include <stdio.h>
#include "utils.h"

void cublas_sgemm(float* a,
                  int lda,
                  float* b,
                  int ldb,
                  float* c,
                  int ldc,
                  int m,
                  int n,
                  int k,
                  U_shape_t us,
                  V_shape_t vs,
                  tiling_info_t ti) {
  // hipPitchedPtr device_M_tensor;
  // hipExtent device_M_tensor_extent = make_hipExtent(
  //     vs.num_tiles * sizeof(float) * us.oc, ti.tile_in_w, ti.tile_in_h);
  // hipMalloc3D(&device_M_tensor, device_M_tensor_extent);
  const float alpha = 1.f;
  const float beta = 0.f;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  auto err=hipblasSgemm(handle,
              transa,
              transb,
              n,
              m,
              k,
              &alpha,
              b,
              ldb,
              a,
              lda,
              &beta,
              c,
              ldc);
  hipDeviceSynchronize();
  hipblasDestroy(handle);
  hipFree(a);
  hipFree(b);
  // printf("%d", err);
}